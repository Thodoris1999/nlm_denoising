#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cassert>

#include "utils.hpp"
#include "shared.hpp"

#define BLOCK_SIZE 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace std;
//using namespace cv;

// denoised_img is the output and should be zero-initialized by the user
__global__ void nlm_kernel(float* denoised_img, float* pad_img, int img_size, float* g_kernel, int w_length, float filt_sigma) {
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int idx = x*img_size + y;
    int pad_length = img_size + w_length - 1;
    int pad_off = w_length/2;

    if (x>=img_size || y>=img_size) return;

    // move gaussian kernel to shared memory
    extern __shared__ float g_kernel_shared[];
    for (int i = 0; i < w_length*w_length / (BLOCK_SIZE*BLOCK_SIZE) + 1; i++) {
        int idx = threadIdx.x + BLOCK_SIZE*threadIdx.y + i * BLOCK_SIZE*BLOCK_SIZE;
        if (idx < w_length*w_length)
            g_kernel_shared[idx] = g_kernel[idx];
    }
    __syncthreads();

    float dist;
    float z = 0;
    denoised_img[idx] = 0;
    for (int i = 0; i < img_size; i++) {
        for (int j = 0; j < img_size; j++) {
            dist = 0;
            for (int k = 0; k < w_length; k++) {
                for (int l = 0; l < w_length; l++) {
                    float val1 = pad_img[(x+k)*pad_length+y+l];
                    float val2 = pad_img[(i+k)*pad_length+j+l];
                    float diff = val1 - val2;
                    int k_idx = k*w_length + l;
                    dist += g_kernel_shared[k_idx]*g_kernel_shared[k_idx]*diff*diff;
                }
            }
            float weight = expf(-dist/filt_sigma);
            z += weight;

            // apply weighted sum to pixel
            denoised_img[idx] += weight*pad_img[(i+pad_off)*pad_length+j+pad_off];
        }
    }
    // normalize
    denoised_img[idx] /= z;
}

float* cuda_shared_non_local_means(float* img,int img_length,int w_length, float filt_sigma, float patch_sigma){
    float *denoised_img = (float*)malloc(img_length*img_length*sizeof(float));
    if(!denoised_img){
        cout << "Couldn't allocate memory for denoised_img in non_local_means\n";
    }
    float* g_kernel = gaussian_kernel(w_length,patch_sigma);
    float* pad_img = padded_image(img,w_length,img_length);

    float* dg_kernel;
    float* dpad_img;
    float* ddenoised_img;
    gpuErrchk( hipMalloc(&dg_kernel, w_length*w_length*sizeof(float)) );
    int pad_length = img_length + w_length - 1;
    gpuErrchk( hipMalloc(&dpad_img, pad_length*pad_length*sizeof(float)) );
    gpuErrchk( hipMalloc(&ddenoised_img, img_length*img_length*sizeof(float)) );

    // move to device
    gpuErrchk( hipMemcpy(dg_kernel, g_kernel, w_length*w_length*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dpad_img, pad_img, pad_length*pad_length*sizeof(float), hipMemcpyHostToDevice) );

    // compute weights and apply NLM
    dim3 blockdim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 griddim((img_length+1)/BLOCK_SIZE, (img_length+1)/BLOCK_SIZE);
    nlm_kernel<<<griddim, blockdim, w_length*w_length*sizeof(float)>>>(ddenoised_img, dpad_img, img_length, dg_kernel, w_length, filt_sigma);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // move to host
    gpuErrchk( hipMemcpy(denoised_img, ddenoised_img, img_length*img_length*sizeof(float), hipMemcpyDeviceToHost) );

    //free allocated memory
    hipFree(dg_kernel);
    hipFree(dpad_img);
    hipFree(ddenoised_img);

    free(g_kernel);
    free(pad_img);

    return denoised_img;
}

