#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cassert>

#include "utils.hpp"

#define BLOCK_SIZE 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace std;
using namespace cv;

// denoised_img is the output and should be zero-initialized by the user
__global__ void nlm_kernel(float* denoised_img, float* pad_img, int img_size, float* g_kernel, int w_length, float filt_sigma) {
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;
    int idx = x*img_size + y;
    int pad_length = img_size + w_length - 1;
    int pad_off = w_length/2;

    if (x>=img_size || y>=img_size) return;

    float dist;
    float z = 0;
    denoised_img[idx] = 0;
    for (int i = 0; i < img_size; i++) {
        for (int j = 0; j < img_size; j++) {
            dist = 0;
            for (int k = 0; k < w_length; k++) {
                for (int l = 0; l < w_length; l++) {
                    float val1 = pad_img[(x+k)*pad_length+y+l];
                    float val2 = pad_img[(i+k)*pad_length+j+l];
                    int k_idx = k*w_length + l;
                    dist += g_kernel[k_idx]*g_kernel[k_idx]*(val1-val2)*(val1-val2);
                }
            }
            float weight = expf(-dist/filt_sigma);
            z += weight;

            // apply weighted sum to pixel
            denoised_img[idx] += weight*pad_img[(i+pad_off)*pad_length+j+pad_off];
        }
    }
    // normalize
    denoised_img[idx] /= z;
}

float* cuda_non_local_means(float* img,int img_length,int w_length, float filt_sigma, float patch_sigma){
    float *denoised_img = (float*)malloc(img_length*img_length*sizeof(float));
    if(!denoised_img){
        cout << "Couldn't allocate memory for denoised_img in non_local_means\n";
    }
    float* g_kernel = gaussian_kernel(w_length,patch_sigma);
    float* pad_img = padded_image(img,w_length,img_length);

    float* dg_kernel;
    float* dpad_img;
    float* ddenoised_img;
    gpuErrchk( hipMalloc(&dg_kernel, w_length*w_length*sizeof(float)) );
    int pad_length = img_length + w_length - 1;
    gpuErrchk( hipMalloc(&dpad_img, pad_length*pad_length*sizeof(float)) );
    gpuErrchk( hipMalloc(&ddenoised_img, img_length*img_length*sizeof(float)) );

    // move to device
    gpuErrchk( hipMemcpy(dg_kernel, g_kernel, w_length*w_length*sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dpad_img, pad_img, pad_length*pad_length*sizeof(float), hipMemcpyHostToDevice) );

    // compute weights and apply NLM
    dim3 blockdim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 griddim((img_length+1)/BLOCK_SIZE, (img_length+1)/BLOCK_SIZE);
    nlm_kernel<<<griddim, blockdim>>>(ddenoised_img, dpad_img, img_length, dg_kernel, w_length, filt_sigma);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // move to host
    gpuErrchk( hipMemcpy(denoised_img, ddenoised_img, img_length*img_length*sizeof(float), hipMemcpyDeviceToHost) );
    /*for (int i = 0; i < img_length; i++) {
        for (int j = 0; j < img_length; j++) {
            std::cout << denoised_img[i+j*img_length] << " ";
        }
        std::cout << std::endl;
    }*/

    //free allocated memory
    hipFree(dg_kernel);
    hipFree(dpad_img);
    hipFree(ddenoised_img);

    free(g_kernel);
    free(pad_img);

    return denoised_img;
}

int main(int argc, char** argv) {
    assert(argc == 2);

    int img_length;
    int img_height;

    int w_length = 5;
    float patch_sigma = 5/3.0;
    float filt_sigma = 0.02;
    float img_noise_stdev = 0.05;

    //initial image
    float* init_img = read_image(argv[1], &img_height, &img_length);   
    if(img_length!=img_height){
        cout << "Not a square image\n";
        exit(-1);
    }   
    show_image(init_img, img_height, img_length);

    //image with noise
    float* noisy_img = (float*) malloc(img_length*img_length*sizeof(float));
    if(!noisy_img){
        cout << "Couldn't allocate memory for noisy_img in main\n";
    }   
    array_add_noise_gauss(init_img, noisy_img, img_noise_stdev, img_length*img_length);
    show_image(noisy_img, img_length, img_length);

    //start timer
    struct timespec init;
    clock_gettime(CLOCK_MONOTONIC, &init);

    // do denoising
    float* denoised_img = cuda_non_local_means(noisy_img,img_length,w_length,filt_sigma,patch_sigma);

    //end timer
    struct timespec last;
    clock_gettime(CLOCK_MONOTONIC, &last);

    long ns;
    uint32_t seconds;
    if(last.tv_nsec <init.tv_nsec){
        ns=init.tv_nsec - last.tv_nsec;
        seconds= last.tv_sec - init.tv_sec -1;
    }

    if(last.tv_nsec >init.tv_nsec){
        ns= last.tv_nsec -init.tv_nsec ;
        seconds= last.tv_sec - init.tv_sec ;
    }
    printf("Image size: %dx%d, patch size: %dx%d\n",img_length,img_length,w_length,w_length);
    printf("Seconds elapsed are %u and the nanoseconds are %ld\n",seconds, ns);

    show_image(denoised_img,img_length,img_length);

    //remainder
    float* diff_img = (float*) malloc(img_length*img_length*sizeof(float));
    if(!diff_img){
        cout << "Couldn't allocate memory for diff_img in main\n";
    }
    array_subtract(noisy_img, denoised_img, diff_img, img_length*img_length);
    show_image(diff_img, img_length, img_length);

    free(init_img);
    free(noisy_img);
    free(denoised_img);
    free(diff_img);
}
