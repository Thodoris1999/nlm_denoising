#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <cassert>

#include "utils.hpp"

using namespace std;
using namespace cv;

float* cuda_non_local_means(float* img,int img_length,int w_length, float filt_sigma, float patch_sigma){
    float *denoised_img = (float*)calloc(img_length*img_length,sizeof(float));
    if(!denoised_img){
        cout << "Couldn't allocate memory for denoised_img in non_local_means\n";
    }

    float* g_kernel = gaussian_kernel(w_length,patch_sigma);
    float* pad_img = padded_image(img,w_length,img_length);

    // find neighborhoods

    // compute weights and apply NLM

    //free allocated memory
    free(g_kernel);
    free(pad_img);
    for(int i=0;i<img_length*img_length;++i){
    //    free(neighborhoods[i]);
    }
    //free(neighborhoods);

    return denoised_img;
}

int main(int argc, char** argv) {
    assert(argc == 2);

    int img_length;
    int img_height;

    int w_length = 5;
    float patch_sigma = 5/3.0;
    float filt_sigma = 0.02;
    float img_noise_stdev = 0.05;

    //initial image
    float* init_img = read_image(argv[1], &img_height, &img_length);   
    if(img_length!=img_height){
        cout << "Not a square image\n";
        exit(-1);
    }   
    show_image(init_img, img_height, img_length);

    //image with noise
    float* noisy_img = (float*) malloc(img_length*img_length*sizeof(float));
    if(!noisy_img){
        cout << "Couldn't allocate memory for noisy_img in main\n";
    }   
    array_add_noise_gauss(init_img, noisy_img, img_noise_stdev, img_length*img_length);
    show_image(noisy_img, img_length, img_length);

    //start timer
    struct timespec init;
    clock_gettime(CLOCK_MONOTONIC, &init);

    // do denoising
    float* denoised_img = cuda_non_local_means(noisy_img,img_length,w_length,filt_sigma,patch_sigma);

    //end timer
    struct timespec last;
    clock_gettime(CLOCK_MONOTONIC, &last);

    long ns;
    uint32_t seconds;
    if(last.tv_nsec <init.tv_nsec){
        ns=init.tv_nsec - last.tv_nsec;
        seconds= last.tv_sec - init.tv_sec -1;
    }

    if(last.tv_nsec >init.tv_nsec){
        ns= last.tv_nsec -init.tv_nsec ;
        seconds= last.tv_sec - init.tv_sec ;
    }
    printf("Image size: %dx%d, patch size: %dx%d\n",img_length,img_length,w_length,w_length);
    printf("Seconds elapsed are %u and the nanoseconds are %ld\n",seconds, ns);

    show_image(denoised_img,img_length,img_length);

    //remainder
    float* diff_img = (float*) malloc(img_length*img_length*sizeof(float));
    if(!diff_img){
        cout << "Couldn't allocate memory for diff_img in main\n";
    }
    array_subtract(noisy_img, denoised_img, diff_img, img_length*img_length);
    show_image(diff_img, img_length, img_length);

    free(init_img);
    free(noisy_img);
    free(denoised_img);
    free(diff_img);
}
