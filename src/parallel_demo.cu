#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include <cassert>

#include "utils.hpp"
#include "parallel.hpp"

using namespace std;

int main(int argc, char** argv) {
    assert(argc == 2);

    int img_length;
    int img_height;

    int w_length = 5;
    float patch_sigma = 5/3.0;
    float img_noise_stdev = 0.05;
    float filt_sigma = 0.02;

    //initial image
    float* init_img = read_image(argv[1], &img_height, &img_length);
    if(img_length!=img_height){
        cout << "Not a square image\n";
        exit(-1);
    }
    show_image(init_img, img_height, img_length);

    //image with noise
    float* noisy_img = (float*) malloc(img_length*img_length*sizeof(float));
    if(!noisy_img){
        cout << "Couldn't allocate memory for noisy_img in main\n";
    }
    array_add_noise_gauss(init_img, noisy_img, img_noise_stdev, img_length*img_length);
    show_image(noisy_img, img_length, img_length);

    double init_img_error = array_rms_error(noisy_img, init_img, img_length);
    printf("Noisy image RMS error: %lf\n", init_img_error);

    //start timer
    struct timespec init;
    clock_gettime(CLOCK_MONOTONIC, &init);

    // do denoising
    float* denoised_img = cuda_non_local_means(noisy_img,img_length,w_length,filt_sigma,patch_sigma);

    //end timer
    struct timespec last;
    clock_gettime(CLOCK_MONOTONIC, &last);

    struct timespec dur = get_duration(init, last);
    double dur_double = timespec2double(dur);

    double denoised_img_error = array_rms_error(noisy_img, denoised_img, img_length);
    printf("Denoised image RMS error: %lf\n", denoised_img_error);

    printf("Image size: %dx%d, patch size: %dx%d\n",img_length,img_length,w_length,w_length);
    printf("Seconds elapsed: %lf\n", dur_double);

    /*float* denoised_img_serial = non_local_means(noisy_img,img_length,w_length,filt_sigma,patch_sigma);
    double denoised_img_error_serial = array_rms_error(noisy_img, denoised_img_serial, img_length);
    int diff = array_compare(denoised_img_serial, denoised_img, img_length);
    printf("Denoised image (serial method) RMS error: %lf\n", denoised_img_error_serial);
    printf("Denoised image serial-parallel different elements: %d\n", diff);*/

    show_image(denoised_img,img_length,img_length);

    // compare with serial implementation

    //remainder
    float* diff_img = (float*) malloc(img_length*img_length*sizeof(float));
    if(!diff_img){
        cout << "Couldn't allocate memory for diff_img in main\n";
    }   
    array_subtract(noisy_img, denoised_img, diff_img, img_length*img_length);
    show_image(diff_img, img_length, img_length);

    free(init_img);
    free(noisy_img);
    free(denoised_img);
    free(diff_img);
}
