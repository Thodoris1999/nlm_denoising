#include "hip/hip_runtime.h"

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <cassert>
#include <sstream>

#include "utils.hpp"

#include "shared.hpp"

using namespace std;
//using namespace cv;

int main(int argc, char* argv[]){
    assert(argc > 2);
    bool visualize = false;
    if (argc > 3 && strcmp(argv[3], "--show") == 0) {
        visualize = true;
    }

    int img_length;
    int img_height;

    int w_length = atoi(argv[2]);
    float patch_sigma = 5/3.0;
    float filt_sigma = 0.02;
    float img_noise_stdev = 0.05;

    //initial image
    float* init_img = read_image(argv[1], &img_height, &img_length);
    if(img_length!=img_height){
        cout << "Not a square image\n";
        exit(-1);
    }
    if (visualize) show_image(init_img, img_height, img_length);

    // time, RMSE, denoised image
    string imagename = string(argv[1]).substr(0, string(argv[1]).size()-4);
    std::stringstream info_ss;
    info_ss << imagename << "_" << w_length << "_shared_info.txt";
    FILE* fp = fopen(info_ss.str().c_str(), "a+");

    //image with noise
    float* noisy_img = (float*) malloc(img_length*img_length*sizeof(float));
    if(!noisy_img){
        cout << "Couldn't allocate memory for noisy_img in main\n";
    }
    array_add_noise_gauss(init_img, noisy_img, img_noise_stdev, img_length*img_length);
    if (visualize) show_image(noisy_img, img_length, img_length);

    for (int i = 0; i < 5; i++) {
        float h = filt_sigma + (i-2)*0.005;
        //start timer
        struct timespec init;
        clock_gettime(CLOCK_MONOTONIC, &init);

        //apply non local means to remove noise
        float* denoised_img = cuda_shared_non_local_means(noisy_img,img_length,w_length,h,patch_sigma);

        //end timer
        struct timespec last;
        clock_gettime(CLOCK_MONOTONIC, &last);

        struct timespec dur = get_duration(init, last);
        double dur_double = timespec2double(dur);

        printf("Image size: %dx%d, patch size: %dx%d\n",img_length,img_length,w_length,w_length);
        if (i == 0) {
            fprintf(fp, "%lf\n", dur_double);
        }

        double denoised_img_error = array_rms_error(noisy_img, denoised_img, img_length);
        printf("Denoised image RMS error: %lf\n", denoised_img_error);
        fprintf(fp, "%f %lf\n", h, denoised_img_error);

        if (visualize) show_image(denoised_img,img_length,img_length);

        //remainder
        float* diff_img = (float*) malloc(img_length*img_length*sizeof(float));
        if(!diff_img){
            cout << "Couldn't allocate memory for diff_img in main\n";
        }
        array_subtract(noisy_img, denoised_img, diff_img, img_length*img_length);
        if (visualize) show_image(diff_img, img_length, img_length);

        std::stringstream dimg_ss, diffimg_ss;
        dimg_ss << imagename << "_" << h << "_" << w_length << "shared_denoised.txt";
        diffimg_ss << imagename <<  "_" << h << "_" << w_length << "shared_diff.txt";
        write_image(dimg_ss.str().c_str(), denoised_img, img_length);
        write_image(diffimg_ss.str().c_str(), diff_img, img_length);
        std::cout << dimg_ss.str().c_str() << std::endl;

        free(denoised_img);
        free(diff_img);
    }

    fclose(fp);
    free(init_img);
    free(noisy_img);
}
